#include<stdio.h>
#include<stdlib.h>

#include "hip/hip_runtime.h"
#include ""


int main() {
  int nDevice;
  hipGetDeviceCount(&nDevice);
  for (int i = 0; i < nDevice; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Dispositivo : %d\n", i);
    printf("  Nombre : %s\n", prop.name);
    printf("  Frecuencia Reloj : %d KHz\n", prop.memoryClockRate);
    printf("  Ancho del Bus de Memoria : %d bits\n", prop.memoryBusWidth);
    printf("  Ancho de Banda : %f GB/s\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
