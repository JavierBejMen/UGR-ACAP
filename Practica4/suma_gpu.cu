#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void operacion(const float *A, const float *B, float *C, int nElementos)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < nElementos)
  {
    for(int j = 0; j < 1; ++j)
      C[i] = pow(pow(log(5*A[i]*100*B[i]+7*A[i])/0.33, 3), 7);
  }
}

int main(int argc, char **argv)
{
  if(argc<2)
  {
    printf("No se ha introducido el directorio de los vectores\n");
    exit(EXIT_FAILURE);
  }

	/** Lectura **/
  FILE *in_0, *in_1;
  char name_0[50], name_1[50];

  strcpy(name_0, argv[1]);
  strcat(name_0, "input0.raw");
  strcpy(name_1, argv[1]);
  strcat(name_1, "input0.raw");

  if ( (in_0 = fopen(name_0,"r")) == NULL ) {
    printf("Error en lectura %s\n", name_0);
    exit(EXIT_FAILURE);
  }
  if ( (in_1 = fopen(name_1, "r")) == NULL ){
    printf("Error en lectura %s\n", name_1);
    exit(EXIT_FAILURE);
  }

  int nEle0, nEle1, nElementos;
  fscanf(in_0, "%d", &nEle0);
  fscanf(in_1, "%d", &nEle1);
  if (nEle0 != nEle1)
  {
    fprintf(stderr, "Distinto tamaño de vectores\n");
    exit(EXIT_FAILURE);
  }
  nElementos = nEle1;
  size_t size = nElementos * sizeof(float);
  printf("Operacion sobre %d elementos\n", nElementos);

  /** Asignacion memoria **/
  float *input0 = (float *)malloc(size);
  float *input1 = (float *)malloc(size);
  float *output = (float *)malloc(size);

  if (input0 == NULL || input1 == NULL || output == NULL)
  {
    fprintf(stderr, "Error allocando memoria\n");
    exit(EXIT_FAILURE);
  }

  /** Inicializa vectores **/
  float aux;
  for (int i = 0; i < nElementos; ++i)
  {
    fscanf(in_0,"%f",&aux);
    input0[i] = aux;
    fscanf(in_1,"%f",&aux);
    input1[i] = aux;
  }

  /** Asignacion memoria cuda **/
  float *device0 = NULL;
  if (hipMalloc((void **)&device0, size) != hipSuccess)
  {
    fprintf(stderr, "Error allocando device 0\n");
    exit(EXIT_FAILURE);
  }

  float *device1 = NULL;
  if (hipMalloc((void **)&device1, size) != hipSuccess)
  {
    fprintf(stderr, "Error alocando device 1\n");
    exit(EXIT_FAILURE);
  }

  float *deviceOut = NULL;
  if (hipMalloc((void **)&deviceOut, size) != hipSuccess)
  {
    fprintf(stderr, "Error alocando device out\n");
    exit(EXIT_FAILURE);
  }

  clock_t start, end;
  double elapsed;
  start = clock();

  /** Enviar datos a dispositivo **/
  if (hipMemcpy(device0, input0, size, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Error copiando input0 a device0\n");
    exit(EXIT_FAILURE);
  }

  if (hipMemcpy(device1, input1, size, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "Error copiando input1 a device1\n");
    exit(EXIT_FAILURE);
  }

  /** Lanzar kernel **/
  int threadsPerBlock = 256;
  int blocksPerGrid =(nElementos + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel: %d bloques de %d hebras\n", blocksPerGrid, threadsPerBlock);
  operacion<<<blocksPerGrid, threadsPerBlock>>>(device0, device1, deviceOut, nElementos);

  hipError_t err = hipGetLastError();

  if (err != hipSuccess)
  {
    fprintf(stderr, "Error en la ejecucion del kernel CUDA\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  /** Traer resultado de dispositivo **/
  printf("Copy output data from the CUDA device to the host memory\n");
  if (hipMemcpy(output, deviceOut, size, hipMemcpyDeviceToHost) != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector C from device to host!\n");
    exit(EXIT_FAILURE);
  }

  end = clock();
  elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;

  printf("Operacion terminada: %f segundos\n", elapsed);

  /** Liberar memoria de dispositivo **/
  if (hipFree(device0) != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector A!\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(device1) != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector B!\n");
    exit(EXIT_FAILURE);
  }

  if (hipFree(deviceOut) != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector C!\n");
    exit(EXIT_FAILURE);
  }

  /** Escritura resultados **/
  FILE * out;
  char name_out[50];
  strcpy(name_out, argv[1]);
  strcat(name_out, "output_gpu.raw");

  if( ( out = fopen( name_out, "w" ) ) == NULL )
  {
    printf( "No se pudo crear la salida %s\n", name_out);
  }

  for (int i = 0; i < nElementos; i++)
  {
    fprintf(out, "%.5f\n", output[i]);
  }

  /** Liberar y cerrar **/
  free(input0);
  free(input1);
  free(output);

  fclose(in_0);
  fclose(in_1);
  fclose(out);

  /** Archivo para plot **/
  FILE * data;
  data = fopen("data_gpu.dat", "a");
  fprintf(data,"%d %f\n", nElementos,elapsed);
  fclose(data);

  return EXIT_SUCCESS;
}
